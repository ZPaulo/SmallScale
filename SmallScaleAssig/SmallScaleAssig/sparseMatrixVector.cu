#include "hip/hip_runtime.h"
#include "SparseMatrixVector.cuh"
#include ""
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

template <unsigned int blockSize>
__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
	if (blockSize >= 64)
		sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32)
		sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16)
		sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8)
		sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4)
		sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2)
		sdata[tid] += sdata[tid + 1];
}


template <unsigned int blockSize>
__global__ void reduceCSR(const int *JA, const double *AS, const double *x, int *IRP, double *output)
{
	unsigned int blockStart = IRP[blockIdx.x];
	unsigned int blockEnd = IRP[blockIdx.x + 1];
	unsigned int tid = threadIdx.x;
	__shared__ double values_in_row[blockSize];
	values_in_row[tid] = 0.0;

	for (int i = tid + blockStart; i < blockEnd; i += blockSize)
		values_in_row[tid] += AS[i] * x[JA[i]];

	__syncthreads();



	if (blockSize >= 512) {
		if (tid < 256) {
			values_in_row[tid] += values_in_row[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			values_in_row[tid] += values_in_row[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			values_in_row[tid] += values_in_row[tid + 64];
		}
		__syncthreads();
	}

	if (tid < 32)
		warpReduce<blockSize>(values_in_row, tid);

	// write result for this block to global mem
	if (tid == 0) {
		output[blockIdx.x] = values_in_row[0];
	}
}

template <unsigned int blockSize>
__global__ void reduceELL(const int *JA, const double *AS, const double *x, unsigned int maxNZ, double *output) {

	unsigned int tid = threadIdx.x;
	__shared__ double values_in_row[blockSize];

	unsigned int end = maxNZ * blockIdx.x + maxNZ;
	unsigned int index;
	values_in_row[tid] = 0.0;

	for (index = blockIdx.x * maxNZ + tid; index < end; index += blockSize)
		values_in_row[tid] += AS[index] * x[JA[index]];

	__syncthreads();

	if (blockSize >= 512) {
		if (tid < 256) {
			values_in_row[tid] += values_in_row[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			values_in_row[tid] += values_in_row[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			values_in_row[tid] += values_in_row[tid + 64];
		}
		__syncthreads();
	}

	if (tid < 32)
		warpReduce<blockSize>(values_in_row, tid);

	// write result for this block to global mem
	if (tid == 0) {
		output[blockIdx.x] = values_in_row[0];
	}
}

double * matrixVectorCSRCUDA(const int *IRP, const int *JA, const double *AS, const double *x, int M, const int N, const int nz, const int numRuns, double *average)
{
	//Host memory init 937807294
	double *result = (double *)malloc(M * sizeof(double));

	//Device memory init
	double *d_AS, *d_x, *d_y;
	int  *d_IRP, *d_JA;
	hipMalloc((void**)&d_IRP, (M + 1) * sizeof(int));
	hipMalloc((void**)&d_JA, nz * sizeof(int));
	hipMalloc((void**)&d_AS, nz * sizeof(double));
	hipMalloc((void**)&d_x, N * sizeof(double));
	hipMalloc((void**)&d_y, M * sizeof(double));

	hipMemcpy(d_IRP, IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_JA, JA, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_AS, AS, nz * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

	const int THREADS_PER_BLOCK = 32;

	float time;
	double myAverage = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for (int i = 0; i < numRuns; i++) {
		hipEventRecord(start, 0);
		reduceCSR<THREADS_PER_BLOCK> << <M, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double) >> > (d_JA, d_AS, d_x, d_IRP, d_y);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		myAverage += time;
	}
	myAverage /= (double)numRuns;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	*average = myAverage;

	hipMemcpy(result, d_y, M * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_IRP);
	hipFree(d_JA);
	hipFree(d_AS);
	hipFree(d_x);
	hipFree(d_y);

	return result;
}

double * matrixVectorELLCUDA(const int maxNZ, const int *JA, const double *AS, const double *x, int M, const int N, const int nz, const int numRuns, double *average)
{
	//Host memory init 937807294
	double *result = (double *)malloc(M * sizeof(double));

	//Device memory init
	double *d_AS, *d_x, *d_y;
	int *d_JA;
	hipMalloc((void**)&d_JA, M * maxNZ * sizeof(int));
	hipMalloc((void**)&d_AS, M * maxNZ * sizeof(double));
	hipMalloc((void**)&d_x, N * sizeof(double));
	hipMalloc((void**)&d_y, M * sizeof(double));

	hipMemcpy(d_JA, JA, M * maxNZ * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_AS, AS, M * maxNZ * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

	const unsigned int THREADS_PER_BLOCK = 32;

	float time;
	double myAverage = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for (int i = 0; i < numRuns; i++) {
		hipEventRecord(start, 0);
		reduceELL<THREADS_PER_BLOCK> << <M, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double) >> > (d_JA, d_AS, d_x, maxNZ, d_y);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		myAverage += time;
	}
	myAverage /= (double)numRuns;
	hipEventDestroy(start);
	hipEventDestroy(stop);
	*average = myAverage;

	hipMemcpy(result, d_y, M * sizeof(double), hipMemcpyDeviceToHost);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.


	hipFree(d_JA);
	hipFree(d_AS);
	hipFree(d_x);
	hipFree(d_y);

	return result;
}